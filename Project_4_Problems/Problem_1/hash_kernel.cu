
#include <hip/hip_runtime.h>
#define MAX     123123123

/* Hash Kernel --------------------------------------
*       Generates an array of hash values from nonces.
*/
__global__
void hash_kernel(unsigned int* hash_array, unsigned int* nonce, unsigned int array_size, unsigned int* transactions, unsigned int n_transactions, unsigned int mod) {

    // Calculate thread index
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < array_size) {
        unsigned int hash = (nonce[index] + transactions[0] * (index + 1)) % mod;
        for(int j = 1; j < n_transactions; j++){
            hash = (hash + transactions[j] * (index + 1)) % mod;
        }
        hash_array[index] = hash;
    }

} // End Hash Kernel //
