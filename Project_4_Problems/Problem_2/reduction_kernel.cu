#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/driver_types.h>
#include <hiprand/hiprand.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstdio>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024
#define MAX     123123123

__global__
void reduction_kernel(unsigned int* hash_array, unsigned int* nonce_array, unsigned int* out_hash, unsigned int* out_nonce, unsigned int array_size) {
    int index = 2 * blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ unsigned int hash_reduction[BLOCK_SIZE];
    __shared__ unsigned int nonce_reduction[BLOCK_SIZE];

    if (index < array_size) {
        hash_reduction[threadIdx.x] = hash_array[index];
        nonce_reduction[threadIdx.x] = nonce_array[index];
    } else {
        // printf("DEBUG: %d >= array_size\n", threadIdx.x);
        hash_reduction[threadIdx.x] = MAX;
        nonce_reduction[threadIdx.x] = MAX;
    }

    if ((index + BLOCK_SIZE) < array_size) {
        if (hash_array[index + BLOCK_SIZE] < hash_reduction[threadIdx.x] && hash_array[index + BLOCK_SIZE] != 0) {
            printf("DEBUG: %d < %d\n", hash_array[index + BLOCK_SIZE], hash_reduction[threadIdx.x]);
            hash_reduction[threadIdx.x] = hash_array[index + BLOCK_SIZE];
            nonce_reduction[threadIdx.x] = nonce_array[index + BLOCK_SIZE];
        }
    }

    for (int stride = 1; stride < BLOCK_SIZE; stride *= 2) {
        __syncthreads();
        if (threadIdx.x % (2*stride) == 0) {
            hash_reduction[threadIdx.x] = hash_reduction[threadIdx.x + stride];
            nonce_reduction[threadIdx.x] = nonce_reduction[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0) {
        *out_hash = hash_reduction[0];
        *out_nonce = nonce_reduction[0];
    }
}
